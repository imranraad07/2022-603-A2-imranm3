#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <tuple>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif

#define THREADS_PER_BLOCK 16

using namespace std;


// help: https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf

__device__ float distance(float * a, int idx_a, int idx_b, int size) {
    float sum = 0;

    for (int i = 0; i < size - 1; i++) {
        float diff = * (a + idx_a + i) - * (a +  idx_b + i);
        sum += diff * diff;
    }

    return sum;
}


__global__ void KNN(float * test, int * predictions, int train_size, int test_size, int k, int num_attributes, int num_classes) {
    // Implements a sequential kNN where for each candidate query an in-place priority queue is maintained to identify the kNN's.

    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row    = blockIdx.y * blockDim.y + threadIdx.y;
    int queryIndex = row * THREADS_PER_BLOCK + column;
    // printf("%d %d %d\n", row, column, queryIndex);
    // int queryIndex = stream * numberElementsPerStream + tid;

    if (queryIndex < test_size) {
        // printf("%d %d %d\n", row, column, queryIndex);

        // Stores k-NN candidates for a query vector as a sorted 2d array. First element is inner product, second is class.
        float candidates[11];
        // Stores bincounts of each class over the final set of candidate NN
        int class_counts[11];

        for (int i = 0; i < 11; i++){
            candidates[i] = FLT_MAX;
            class_counts[i] = 0;
        }

        __syncthreads();

        for (int keyIndex = 0; keyIndex < train_size; keyIndex ++) {
            float dist = distance(test, queryIndex * num_attributes, num_attributes * test_size + keyIndex * num_attributes, num_attributes);
            __syncthreads();


            // Add to our candidates
            for (int c = 0; c < k; c++) {
                if (dist < candidates[2 * c]) {
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for (int x = k - 2; x >= c; x--) {
                        candidates[2 * x + 2] = candidates[2 * x];
                        candidates[2 * x + 3] = candidates[2 * x + 1];
                    }
                    // Set key vector as potential k NN
                    candidates[2 * c] = dist;
                    candidates[2 * c + 1] = test[num_attributes * test_size + keyIndex * num_attributes + num_attributes - 1]; // class value
                    break;
                }
            }
        }
        __syncthreads();


        // Bincount the candidate labels and pick the most common
        for (int i = 0; i < k; i++) {
            class_counts[ (int) candidates[2 * i + 1]] ++;
        }

        int max = -1;
        int max_index = 0;
        for (int i = 0; i < num_classes; i++) {
            if (class_counts[ i] > max) {
                max = class_counts[i];
                max_index = i;
            }
        }
        predictions[queryIndex] = max_index;
    }
}

int * computeConfusionMatrix(int * predictions, ArffData * dataset) {
    int * confusionMatrix = (int * ) calloc(dataset -> num_classes() * dataset -> num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses

    for (int i = 0; i < dataset -> num_instances(); i++) // for each instance compare the true class and predicted class
    {
        int trueClass = dataset -> get_instance(i) -> get(dataset -> num_attributes() - 1) -> operator int32();
        int predictedClass = predictions[i];

        confusionMatrix[trueClass * dataset -> num_classes() + predictedClass]++;
    }

    return confusionMatrix;
}

float computeAccuracy(int * confusionMatrix, ArffData * dataset) {
    int successfulPredictions = 0;

    for (int i = 0; i < dataset -> num_classes(); i++) {
        successfulPredictions += confusionMatrix[i * dataset -> num_classes() + i]; // elements in the diagonal are correct predictions
    }

    return successfulPredictions / (float) dataset -> num_instances();
}

int main(int argc, char * argv[]) {
    if (argc != 4) {
        cout << "Usage: ./main datasets/trainfile.arff datasets/testfile.arff k" << endl;
        exit(0);
    }

    int k = strtol(argv[3], NULL, 10);

    // Open the datasets
    ArffParser parserTrain(argv[1]);
    ArffParser parserTest(argv[2]);
    ArffData * train = parserTrain.parse();
    ArffData * test = parserTest.parse();

    int num_attributes = train -> num_attributes();
    int num_classes = train -> num_classes();
    int train_size = (int) train -> num_instances();
    int test_size = (int) test -> num_instances();

    printf("number of attributes: %d\n", num_attributes);
    printf("number of classes: %d\n", num_classes);
    printf("number of train instances: %d\n", train_size);
    printf("number of test instances: %d\n", test_size);


    float * h_test_instances = (float * ) malloc((train_size + test_size) * num_attributes * sizeof(float));
    for (int i = 0; i < test_size; i++) {
        for (int j = 0; j < num_attributes; j++) {
            h_test_instances[i * num_attributes + j] = test -> get_instance(i) -> get(j) -> operator float();
        }
    }
    for (int i = 0; i < train_size; i++) {
        for (int j = 0; j < num_attributes; j++) {
            h_test_instances[ (test_size + i) * num_attributes + j] = train -> get_instance(i) -> get(j) -> operator float();
        }
    }

    // Predictions is the array where you have to return the class predicted (integer) for the test dataset instances
    int * h_predictions = (int * ) malloc(test_size * sizeof(int));


    // Allocate device memory
    float * d_test_instances;
    int * d_predictions;

    hipMalloc( & d_test_instances, (test_size + train_size) * num_attributes * sizeof(float));
    hipMalloc( & d_predictions, test_size * sizeof(int));

    // cuda timing: https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // Copy host memory to device memory
    hipMemcpy(d_test_instances, h_test_instances, (train_size + test_size) * num_attributes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_predictions, h_predictions, test_size * sizeof(int), hipMemcpyHostToDevice);

    // Configure the block and grid sizes
    int gridDim = (test_size + THREADS_PER_BLOCK - 1) / (THREADS_PER_BLOCK);

    dim3 blockSize (THREADS_PER_BLOCK, 1);
    dim3 gridSize (gridDim, 1);

    hipEventRecord(start);
    KNN <<< gridSize, blockSize >>> (d_test_instances, d_predictions, train_size, test_size, k, num_attributes, num_classes);
    hipDeviceSynchronize();                                                                                                                                                    

    // Transfer device results to host memory
    hipMemcpy(h_predictions, d_predictions, test_size * sizeof(int), hipMemcpyDeviceToHost);

    hipError_t hipError_t = hipGetLastError();

    if (hipError_t != hipSuccess) {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Compute the confusion matrix
    int * confusionMatrix = computeConfusionMatrix(h_predictions, test);
    // Calculate the accuracy
    float accuracy = computeAccuracy(confusionMatrix, test);

    printf("The %i-NN classifier for %d test instances on %d train instances required %f ms CPU time. Accuracy was %.2f%%\n", k, test_size, train_size, milliseconds, (accuracy * 100));

    // Free device global memory
    hipFree(d_test_instances);
    hipFree(d_predictions);

    // Free host memory
    free(h_test_instances);
    free(h_predictions);

    return 0;
}